// Copyright 2021 Bryan Baker

// Compile using:
// nvcc Ch2_vecadd.cu -o vecadd


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float *h_A, float *h_B, float *h_C, int n) {
  for(int i = 0; i < n; i++) h_C[i] = h_A[i] + h_B[i];
}

double wctime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main() {
  float *A, *B, *C, *d_A, *d_B, *d_C;
  int i, N=10000000;
  double t1;
  float nops;

  A = (float*)malloc(N*sizeof(float));
  B = (float*)malloc(N*sizeof(float));
  C = (float*)malloc(N*sizeof(float));
  hipMalloc((void**)&d_A, N*sizeof(float));
  hipMalloc((void**)&d_B, N*sizeof(float));
  hipMalloc((void**)&d_C, N*sizeof(float));

  for(i=0; i<N; i++) {
    A[i] = (float) rand() / (float) rand();
    B[i] = (float) rand() / (float) rand();
  }
  for(i=0; i<N; i++)
    C[i] = 0.0;

  t1 = wctime();
  vecAdd(A, B, C, N);
  t1 = wctime() - t1;

  /*printf("Vector A sample: ");
  for(i=0; i<10; i++)
    printf("%0.3f ", A[i]);
  printf("\n");
  printf("Vector B sample: ");
  for(i=0; i<10; i++)
    printf("%0.3f ", B[i]);
  printf("\n");
  printf("Vector C sample: ");
  for(i=0; i<10; i++)
    printf("%0.3f ", C[i]);
  printf("\n");*/

  printf("CPU:\n");
  printf("Finished in %lf seconds.\n", t1);
  t1 *= (1.E+09);
  nops = (float) N;
  printf("Performance = %f GFLOPs\n", nops/t1);
  printf("\n");

  for(i=0; i<N; i++)
    C[i] = 0.0;

  t1 = wctime();
  hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, N*sizeof(float), hipMemcpyHostToDevice);

  vecAddKernel<<<ceil(N/1024.0), 1024>>>(d_A, d_B, d_C, N);

  hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
  t1 = wctime() - t1;

  printf("Cuda with data transfer:\n");
  printf("Finished in %lf seconds.\n", t1);
  t1 *= (1.E+09);
  nops = (float) N;
  printf("Performance = %f GFLOPs\n", nops/t1);
  printf("\n");

  for(i=0; i<N; i++)
    C[i] = 0.0;

  hipMemcpy(d_C, C, N*sizeof(float), hipMemcpyHostToDevice);

  t1 = wctime();
  vecAddKernel<<<ceil(N/1024.0), 1024>>>(d_A, d_B, d_C, N);
  t1 = wctime() - t1;

  hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);

  printf("Cuda without data transfer:\n");
  printf("Finished in %lf seconds.\n", t1);
  t1 *= (1.E+09);
  nops = (float) N;
  printf("Performance = %f GFLOPs\n", nops/t1);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
}
